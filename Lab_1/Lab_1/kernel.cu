#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include ""
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <chrono>

#define BLOCK_NUM 8


__global__
void cos_gpu(float *arg, float *res) {
	res[threadIdx.x] = cosf(arg[threadIdx.x]);
}

float randFloat(float min, float max) {
	return  (max - min) * ((((float)rand()) / (float)RAND_MAX)) + min;
}

void gpu_compute(int N) {

	float *host_args = new float[N];
	float *host_res = new float[N];

	for (auto i = 0; i < N; i++)
		host_args[i] = randFloat(0, 1);

	float *device_args, *device_res;
	const int size = N * sizeof(float);

	// ������ ������� �������
	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// ��������� ������ ��� ���������� � device
	hipMalloc(&device_args, size);
	hipMalloc(&device_res, size);

	// ������������ ���������� �� host � device
	hipMemcpy(device_args, host_args, size, hipMemcpyHostToDevice);


	// ���������� �������
	cos_gpu <<<BLOCK_NUM, N / BLOCK_NUM >>> (device_args, device_res);

	// ���������� ����������
	hipDeviceSynchronize();

	// ������������ ���������� �� device � host
	hipMemcpy(host_res, device_res, size, hipMemcpyDeviceToHost);

	// ����� ������� �������
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	// ������� ������
	hipFree(device_args);
	hipFree(device_res);
	delete[] host_args;
	delete[] host_res;

	std::cout << "\n\n=====================   GPU   =====================\n";
	std::cout << "DEVICE GPU compute time: " << gpuTime << " milliseconds\n\n";
}

void cpu_compute(int N) {

	float *host_args = new float[N];
	float *host_res = new float[N];

	for (auto i = 0; i < N; i++)
		host_args[i] = randFloat(0, 1);

	using micro = std::chrono::microseconds;
	auto start = std::chrono::high_resolution_clock::now();

	// ���������� �������
	for (auto i = 0; i < N; i++)
		host_res[i] = cosf(host_args[i]);

	auto stop = std::chrono::high_resolution_clock::now();
	auto cpuTime = std::chrono::duration_cast<micro>(stop - start).count() / 1000.0f;

	delete[] host_args;
	delete[] host_res;

	std::cout << "\n\n=====================   CPU   =====================\n";
	std::cout << "HOST CPU compute time: " << cpuTime << " milliseconds\n\n";
}

int main()
{
	int count;

	std::cout << "Enter the number of cosine calculations: ";
	std::cin >> count;

	gpu_compute(count);
	cpu_compute(count);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}
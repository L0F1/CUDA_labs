#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include ""
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <chrono>


__global__
void cos_gpu(float *arg, float *res) {

	int b = blockIdx.x * blockDim.x;
	res[threadIdx.x + b] = cosf(arg[threadIdx.x + b]);
}

float randFloat(float min, float max) {
	srand((unsigned int)time(NULL));
	return  (max - min) * ((((float)rand()) / (float)RAND_MAX)) + min;
}

void gpu_compute(int N) {

	float *host_args = new float[N];
	float *host_res = new float[N];

	for (auto i = 0; i < N; i++)
		host_args[i] = randFloat(0, 1);

	float *device_args, *device_res;
	const int size = N * sizeof(float);

	// ������ ������� �������
	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// ��������� ������ ��� ���������� � device
	hipMalloc(&device_args, size);
	hipMalloc(&device_res, size);

	// ������������ ���������� �� host � device
	hipMemcpy(device_args, host_args, size, hipMemcpyHostToDevice);


	// ���������� �������
	cos_gpu <<<1, N >>> (device_args, device_res);

	// ���������� ����������
	hipDeviceSynchronize();

	// ������������ ���������� �� device � host
	hipMemcpy(host_res, device_res, size, hipMemcpyDeviceToHost);

	// ����� ������� �������
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	// ������� ������
	hipFree(device_args);
	hipFree(device_res);
	delete[] host_args;
	delete[] host_res;

	std::cout << "\n\n=====================   GPU   =====================\n";
	std::cout << "DEVICE GPU compute time: " << gpuTime * 1000 << " microseconds\n\n";
}

void cpu_compute(int N) {

	float *host_args = new float[N];
	float *host_res = new float[N];

	for (auto i = 0; i < N; i++)
		host_args[i] = randFloat(0, 1);

	using micro = std::chrono::microseconds;
	auto start = std::chrono::high_resolution_clock::now();

	// ���������� �������
	for (auto i = 0; i < N; i++)
		host_res[i] = cosf(host_args[i]);

	auto stop = std::chrono::high_resolution_clock::now();
	auto cpuTime = std::chrono::duration_cast<micro>(stop - start).count();

	delete[] host_args;
	delete[] host_res;

	std::cout << "\n\n=====================   CPU   =====================\n";
	std::cout << "HOST CPU compute time: " << cpuTime << " microseconds\n\n";
}

int main()
{
	int count;

	std::cout << "Enter the number of cosine calculations: ";
	std::cin >> count;

	gpu_compute(count);
	cpu_compute(count);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}